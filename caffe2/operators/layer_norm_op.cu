#include "hip/hip_runtime.h"
#include "caffe2/core/operator_c10wrapper.h"
#include "caffe2/operators/layer_norm_op.h"

#include "caffe2/core/context_gpu.h"
#include "caffe2/utils/math.h"
#include "caffe2/utils/math/reduce.cuh"
#include "caffe2/utils/math/utils.h"

namespace caffe2 {

namespace {

template <typename T>
__global__ void ComputeStdDevAndFusedParamsCUDAKernel(
    const int N,
    const T epsilon,
    const T* mean,
    const T* var,
    T* stddev,
    T* scale,
    T* bias);

template <>
__global__ void ComputeStdDevAndFusedParamsCUDAKernel<float>(
    const int N,
    const float epsilon,
    const float* mean,
    const float* var,
    float* stddev,
    float* scale,
    float* bias) {
  const int index = blockIdx.x * CAFFE_CUDA_NUM_THREADS + threadIdx.x;
  if (index < N) {
#if __CUDA_ARCH__ >= 350
    const float rstd = rsqrtf(__ldg(var + index) + epsilon);
    stddev[index] = rstd * (__ldg(var + index) + epsilon);
    scale[index] = rstd;
    bias[index] = -rstd * __ldg(mean + index);
#else
    const float rstd = rsqrtf(var[index] + epsilon);
    stddev[index] = rstd * (var[index] + epsilon);
    scale[index] = rstd;
    bias[index] = -rstd * mean[index];
#endif
  }
}

template <typename T>
__global__ void LayerNormForwardCUDAKernel(
    const int M,
    const int N,
    const T* X,
    const T* scale,
    const T* bias,
    T* Y);

template <>
__global__ void LayerNormForwardCUDAKernel<float>(
    const int M,
    const int N,
    const float* X,
    const float* scale,
    const float* bias,
    float* Y) {
  const int size = M * N;
  const int index = blockIdx.x * CAFFE_CUDA_NUM_THREADS + threadIdx.x;
  if (index < size) {
    const int i = index / N;
#if __CUDA_ARCH__ >= 350
    Y[index] = fmaf(__ldg(X + index), __ldg(scale + i), __ldg(bias + i));
#else
    Y[index] = fmaf(X[index], scale[i], bias[i]);
#endif
  }
}

template <typename T>
__global__ void ComputeInternalGradientsCUDAKernel(
    const int M,
    const int N,
    const T* dY,
    const T* X,
    T* ds,
    T* db) {
  __shared__ typename BlockReduce<T>::TempStorage ds_storage;
  __shared__ typename BlockReduce<T>::TempStorage db_storage;
  const int i = blockIdx.x;
  T ds_val = 0;
  T db_val = 0;
  for (int j = threadIdx.x; j < N; j += blockDim.x) {
    const int index = i * N + j;
#if __CUDA_ARCH__ >= 350
    ds_val += __ldg(dY + index) * __ldg(X + index);
    db_val += __ldg(dY + index);
#else
    ds_val += dY[index] * X[index];
    db_val += dY[index];
#endif
  }
  ds_val = BlockReduce<T>(ds_storage).Sum(ds_val);
  db_val = BlockReduce<T>(db_storage).Sum(db_val);
  if (threadIdx.x == 0) {
    ds[i] = ds_val;
    db[i] = db_val;
  }
}

template <typename T>
__global__ void ComputeFusedParamsCUDAKernel(
    const int M,
    const int N,
    const T* mean,
    const T* sig,
    const T* ds,
    const T* db,
    T* dY_scale,
    T* X_scale,
    T* bias);

template <>
__global__ void ComputeFusedParamsCUDAKernel<float>(
    const int M,
    const int N,
    const float* mean,
    const float* sig,
    const float* ds,
    const float* db,
    float* dY_scale,
    float* X_scale,
    float* bias) {
  const float scale = 1.0f / static_cast<float>(N);
  const int index = blockIdx.x * CAFFE_CUDA_NUM_THREADS + threadIdx.x;
  if (index < M) {
#if __CUDA_ARCH__ >= 350
    const float rsig = 1.0f / __ldg(sig + index);
    const float X_scale_val =
        fmaf(__ldg(db + index), __ldg(mean + index), -__ldg(ds + index)) *
        math::utils::Cube<float>(rsig) * scale;
    dY_scale[index] = rsig;
    X_scale[index] = X_scale_val;
    bias[index] = -fmaf(
        X_scale_val, __ldg(mean + index), __ldg(db + index) * rsig * scale);
#else
    const float rsig = 1.0f / sig[index];
    const float X_scale_val = fmaf(db[index], mean[index], -ds[index]) *
        math::utils::Cube<float>(rsig) * scale;
    dY_scale[index] = rsig;
    X_scale[index] = X_scale_val;
    bias[index] = -fmaf(X_scale_val, mean[index], db[index] * rsig * scale);
#endif
  }
}

template <typename T>
__global__ void LayerNormBackwardCUDAKenrel(
    const int M,
    const int N,
    const T* dY_scale,
    const T* dY,
    const T* X_scale,
    const T* X,
    const T* bias,
    T* dX);

template <>
__global__ void LayerNormBackwardCUDAKenrel<float>(
    const int M,
    const int N,
    const float* dY_scale,
    const float* dY,
    const float* X_scale,
    const float* X,
    const float* bias,
    float* dX) {
  const int size = M * N;
  const int index = blockIdx.x * CAFFE_CUDA_NUM_THREADS + threadIdx.x;
  if (index < size) {
    const int i = index / N;
#if __CUDA_ARCH__ >= 350
    dX[index] = fmaf(
        __ldg(dY + index),
        __ldg(dY_scale + i),
        fmaf(__ldg(X + index), __ldg(X_scale + i), __ldg(bias + i)));
#else
    dX[index] =
        fmaf(dY[index], dY_scale[i], fmaf(X[index], X_scale[i], bias[i]));
#endif
  }
}

} //  namespace

template <>
template <typename T>
void LayerNormOp<HIPContext>::ComputeStdDevAndFusedParams(
    const int N,
    const T* mean,
    const T* var,
    T* stddev,
    T* scale,
    T* bias,
    float epsilon,
    HIPContext* context) {
  const int M = math::DivUp(N, CAFFE_CUDA_NUM_THREADS);
  ComputeStdDevAndFusedParamsCUDAKernel<T>
      <<<M, CAFFE_CUDA_NUM_THREADS, 0, context->hip_stream()>>>(
          N, static_cast<T>(epsilon), mean, var, stddev, scale, bias);
}

template <>
template <typename T>
void LayerNormOp<HIPContext>::LayerNormForward(
    const int M,
    const int N,
    const T* X,
    const T* scale,
    const T* bias,
    T* Y,
    HIPContext* context) {
  const int K = math::DivUp(M * N, CAFFE_CUDA_NUM_THREADS);
  LayerNormForwardCUDAKernel<T>
      <<<K, CAFFE_CUDA_NUM_THREADS, 0, context->hip_stream()>>>(
          M, N, X, scale, bias, Y);
}

REGISTER_HIP_OPERATOR(LayerNorm, LayerNormOp<HIPContext>);

template <>
template <typename T>
void LayerNormGradientOp<HIPContext>::ComputeInternalGradients(
    const int M,
    const int N,
    const T* dY,
    const T* X,
    T* ds,
    T* db) {
  ComputeInternalGradientsCUDAKernel<T>
      <<<M, CAFFE_CUDA_NUM_THREADS, 0, context_.hip_stream()>>>(
          M, N, dY, X, ds, db);
}

template <>
template <typename T>
void LayerNormGradientOp<HIPContext>::ComputeFusedParams(
    const int M,
    const int N,
    const T* mean,
    const T* sig,
    const T* ds,
    const T* db,
    T* dY_scale,
    T* X_scale,
    T* bias) {
  const int K = math::DivUp(M, CAFFE_CUDA_NUM_THREADS);
  ComputeFusedParamsCUDAKernel<T>
      <<<K, CAFFE_CUDA_NUM_THREADS, 0, context_.hip_stream()>>>(
          M, N, mean, sig, ds, db, dY_scale, X_scale, bias);
}

template <>
template <typename T>
void LayerNormGradientOp<HIPContext>::LayerNormBackward(
    const int M,
    const int N,
    const T* dY_scale,
    const T* dY,
    const T* X_scale,
    const T* X,
    const T* bias,
    T* dX) {
  const int K = math::DivUp(M * N, CAFFE_CUDA_NUM_THREADS);
  LayerNormBackwardCUDAKenrel<T>
      <<<K, CAFFE_CUDA_NUM_THREADS, 0, context_.hip_stream()>>>(
          M, N, dY_scale, dY, X_scale, X, bias, dX);
}

REGISTER_HIP_OPERATOR(LayerNormGradient, LayerNormGradientOp<HIPContext>);

} // namespace caffe2

C10_REGISTER_CAFFE2_OPERATOR_CUDA(
    LayerNorm,
    caffe2::LayerNormOp<caffe2::HIPContext>)

namespace caffe2 {
REGISTER_C10_OPERATOR_FOR_CAFFE2_DISPATCH_CUDA(
    "_caffe2::LayerNorm",
    C10LayerNorm_DontUseThisOpYet);
}
